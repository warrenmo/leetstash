#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M && col < K) {
        float total = 0.f;
        for (int i = 0; i < N; ++i) {
            total += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = total;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
